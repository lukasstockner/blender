#include "hip/hip_runtime.h"
/*
 * Copyright 2011-2015 Blender Foundation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#define __KERNEL_CUDA__
#define __KERNEL_CUDA_SPLIT__
#define __SPLIT_KERNEL__

#include "split/kernel_next_iteration_setup.h"

extern "C" 
__global__ void kernel_cuda_path_trace_next_iteration_setup(
        ccl_global char *sd,                  /* Required for setting up ray for next iteration */
        ccl_global uint *rng_coop,            /* Required for setting up ray for next iteration */
        ccl_global float3 *throughput_coop,   /* Required for setting up ray for next iteration */
        PathRadiance *PathRadiance_coop,      /* Required for setting up ray for next iteration */
        ccl_global Ray *Ray_coop,             /* Required for setting up ray for next iteration */
        ccl_global PathState *PathState_coop, /* Required for setting up ray for next iteration */
        ccl_global Ray *LightRay_dl_coop,     /* Required for radiance update - direct lighting */
        ccl_global int *ISLamp_coop,          /* Required for radiance update - direct lighting */
        ccl_global BsdfEval *BSDFEval_coop,   /* Required for radiance update - direct lighting */
        ccl_global Ray *LightRay_ao_coop,     /* Required for radiance update - AO */
        ccl_global float3 *AOBSDF_coop,       /* Required for radiance update - AO */
        ccl_global float3 *AOAlpha_coop,      /* Required for radiance update - AO */
        ccl_global char *ray_state,           /* Denotes the state of each ray */
        ccl_global int *Queue_data,           /* Queue memory */
        ccl_global int *Queue_index,          /* Tracks the number of elements in each queue */
        int queuesize,                        /* Size (capacity) of each queue */
        ccl_global char *use_queues_flag)     /* flag to decide if scene_intersect kernel should
                                               * use queues to fetch ray index */
{
	ccl_local_var unsigned int local_queue_atomics;
	if(ccl_local_thread_x == 0 && ccl_local_thread_y == 0) {
		local_queue_atomics = 0;
	}
	ccl_local_barrier();

	if(ccl_thread_x == 0 && ccl_thread_y == 0) {
		/* If we are here, then it means that scene-intersect kernel
		* has already been executed atleast once. From the next time,
		* scene-intersect kernel may operate on queues to fetch ray index
		*/
		use_queues_flag[0] = 1;

		/* Mark queue indices of QUEUE_SHADOW_RAY_CAST_AO_RAYS and
		 * QUEUE_SHADOW_RAY_CAST_DL_RAYS queues that were made empty during the
		 * previous kernel.
		 */
		Queue_index[QUEUE_SHADOW_RAY_CAST_AO_RAYS] = 0;
		Queue_index[QUEUE_SHADOW_RAY_CAST_DL_RAYS] = 0;
	}

	char enqueue_flag = 0;
	int ray_index = ccl_thread_y*ccl_size_x + ccl_thread_x;
	ray_index = get_ray_index(ray_index,
	                          QUEUE_ACTIVE_AND_REGENERATED_RAYS,
	                          Queue_data,
	                          queuesize,
	                          0);

	if(ray_index != QUEUE_EMPTY_SLOT) {
		enqueue_flag = kernel_next_iteration_setup(NULL,
		                                           (ShaderData *)sd,
		                                           rng_coop,
		                                           throughput_coop,
		                                           PathRadiance_coop,
		                                           Ray_coop,
		                                           PathState_coop,
		                                           LightRay_dl_coop,
		                                           ISLamp_coop,
		                                           BSDFEval_coop,
		                                           LightRay_ao_coop,
		                                           AOBSDF_coop,
		                                           AOAlpha_coop,
		                                           ray_state,
		                                           use_queues_flag,
		                                           ray_index);
	}

	/* Enqueue RAY_UPDATE_BUFFER rays. */
	enqueue_ray_index_local(ray_index,
	                        QUEUE_HITBG_BUFF_UPDATE_TOREGEN_RAYS,
	                        enqueue_flag,
	                        queuesize,
	                        &local_queue_atomics,
	                        Queue_data,
	                        Queue_index);
}
