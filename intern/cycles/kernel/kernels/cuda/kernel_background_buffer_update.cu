#include "hip/hip_runtime.h"
/*
 * Copyright 2011-2015 Blender Foundation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#define __KERNEL_CUDA__
#define __KERNEL_CUDA_SPLIT__
#define __SPLIT_KERNEL__

#include "split/kernel_background_buffer_update.h"

extern "C" 
__global__ void kernel_cuda_path_trace_background_buffer_update(
        ccl_global char *sd,
        ccl_global float *per_sample_output_buffers,
        ccl_global uint *rng_state,
        ccl_global uint *rng_coop,             /* Required for buffer Update */
        ccl_global float3 *throughput_coop,    /* Required for background hit processing */
        PathRadiance *PathRadiance_coop,       /* Required for background hit processing and buffer Update */
        ccl_global Ray *Ray_coop,              /* Required for background hit processing */
        ccl_global PathState *PathState_coop,  /* Required for background hit processing */
        ccl_global float *L_transparent_coop,  /* Required for background hit processing and buffer Update */
        ccl_global char *ray_state,            /* Stores information on the current state of a ray */
        int sw, int sh, int sx, int sy, int stride,
        int rng_state_offset_x,
        int rng_state_offset_y,
        int rng_state_stride,
        ccl_global unsigned int *work_array,   /* Denotes work of each ray */
        ccl_global int *Queue_data,            /* Queues memory */
        ccl_global int *Queue_index,           /* Tracks the number of elements in each queue */
        int queuesize,                         /* Size (capacity) of each queue */
        int end_sample,
        int start_sample,
        int parallel_samples                  /* Number of samples to be processed in parallel */
#ifdef __WORK_STEALING__
        , ccl_global unsigned int *work_pool_wgs,
        unsigned int num_samples
#endif
#ifdef __KERNEL_DEBUG__
        , DebugData *debugdata_coop
#endif
) {
	ccl_local_var unsigned int local_queue_atomics;
	if(ccl_local_thread_x == 0 && ccl_local_thread_y == 0) {
		local_queue_atomics = 0;
	}
	ccl_local_barrier();

	int ray_index = ccl_thread_y*ccl_size_x + ccl_thread_x;
	if(ray_index == 0) {
		/* We will empty this queue in this kernel. */
		Queue_index[QUEUE_HITBG_BUFF_UPDATE_TOREGEN_RAYS] = 0;
	}
	char enqueue_flag = 0;
	ray_index = get_ray_index(ray_index,
	                          QUEUE_HITBG_BUFF_UPDATE_TOREGEN_RAYS,
	                          Queue_data,
	                          queuesize,
	                          1);

	/* TODO(lukas): Maybe exit earlier? barrier is problematic, though... */

	if(ray_index != QUEUE_EMPTY_SLOT) {
		enqueue_flag =
			kernel_background_buffer_update(NULL,
			                                (ShaderData *)sd,
			                                per_sample_output_buffers,
			                                rng_state,
			                                rng_coop,
			                                throughput_coop,
			                                PathRadiance_coop,
			                                Ray_coop,
			                                PathState_coop,
			                                L_transparent_coop,
			                                ray_state,
			                                sw, sh, sx, sy, stride,
			                                rng_state_offset_x,
			                                rng_state_offset_y,
			                                rng_state_stride,
			                                work_array,
			                                end_sample,
			                                start_sample,
#ifdef __WORK_STEALING__
			                                work_pool_wgs,
			                                num_samples,
#endif
#ifdef __KERNEL_DEBUG__
			                                debugdata_coop,
#endif
			                                parallel_samples,
			                                ray_index);
	}

	/* Enqueue RAY_REGENERATED rays into QUEUE_ACTIVE_AND_REGENERATED_RAYS;
	 * These rays will be made active during next SceneIntersectkernel.
	 */
	enqueue_ray_index_local(ray_index,
	                        QUEUE_ACTIVE_AND_REGENERATED_RAYS,
	                        enqueue_flag,
	                        queuesize,
	                        &local_queue_atomics,
	                        Queue_data,
	                        Queue_index);
}
