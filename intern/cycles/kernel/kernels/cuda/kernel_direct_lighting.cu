#include "hip/hip_runtime.h"
/*
 * Copyright 2011-2015 Blender Foundation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#define __KERNEL_CUDA__
#define __KERNEL_CUDA_SPLIT__
#define __SPLIT_KERNEL__

#include "split/kernel_direct_lighting.h"
#include "kernel_split.cuh"

extern "C" 
SPLIT_KERNEL_BOUNDS
__global__ void kernel_cuda_path_trace_direct_lighting(
        ccl_global char *sd,                    /* Required for direct lighting */
        ccl_global char *sd_DL,                 /* Required for direct lighting */
        ccl_global uint *rng_coop,              /* Required for direct lighting */
        ccl_global PathState *PathState_coop,   /* Required for direct lighting */
        ccl_global int *ISLamp_coop,            /* Required for direct lighting */
        ccl_global Ray *LightRay_coop,          /* Required for direct lighting */
        ccl_global BsdfEval *BSDFEval_coop,     /* Required for direct lighting */
        ccl_global char *ray_state,             /* Denotes the state of each ray */
        ccl_global int *Queue_data,             /* Queue memory */
        ccl_global int *Queue_index,            /* Tracks the number of elements in each queue */
        int queuesize)                          /* Size (capacity) of each queue */
{
	ccl_local_var unsigned int local_queue_atomics;
	if(ccl_local_thread_x == 0 && ccl_local_thread_y == 0) {
		local_queue_atomics = 0;
	}
	ccl_local_barrier();

	char enqueue_flag = 0;
	int ray_index = ccl_thread_y*ccl_size_x + ccl_thread_x;
	ray_index = get_ray_index(ray_index,
	                          QUEUE_ACTIVE_AND_REGENERATED_RAYS,
	                          Queue_data,
	                          queuesize,
	                          0);

	if(ray_index != QUEUE_EMPTY_SLOT) {
		enqueue_flag = kernel_direct_lighting(NULL,
		                                      (ShaderData *)sd,
		                                      (ShaderData *)sd_DL,
		                                      rng_coop,
		                                      PathState_coop,
		                                      ISLamp_coop,
		                                      LightRay_coop,
		                                      BSDFEval_coop,
		                                      ray_state,
		                                      ray_index);

	}

#ifdef __EMISSION__
	/* Enqueue RAY_SHADOW_RAY_CAST_DL rays. */
	enqueue_ray_index_local(ray_index,
	                        QUEUE_SHADOW_RAY_CAST_DL_RAYS,
	                        enqueue_flag,
	                        queuesize,
	                        &local_queue_atomics,
	                        Queue_data,
	                        Queue_index);
#endif
}
