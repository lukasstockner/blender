#include "hip/hip_runtime.h"
/*
 * Copyright 2011-2015 Blender Foundation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#define __KERNEL_CUDA__
#define __KERNEL_CUDA_SPLIT__
#define __SPLIT_KERNEL__

#include "split/kernel_shader_eval.h"
#include "kernel_split.cuh"

extern "C" 
SPLIT_KERNEL_BOUNDS
__global__ void kernel_cuda_path_trace_shader_eval(
        ccl_global char *sd,                   /* Output ShaderData structure to be filled */
        ccl_global uint *rng_coop,             /* Required for rbsdf calculation */
        ccl_global Ray *Ray_coop,              /* Required for setting up shader from ray */
        ccl_global PathState *PathState_coop,  /* Required for all functions in this kernel */
        Intersection *Intersection_coop,       /* Required for setting up shader from ray */
        ccl_global char *ray_state,            /* Denotes the state of each ray */
        ccl_global int *Queue_data,            /* queue memory */
        ccl_global int *Queue_index,           /* Tracks the number of elements in each queue */
        int queuesize)                         /* Size (capacity) of each queue */
{
	/* Enqeueue RAY_TO_REGENERATE rays into QUEUE_HITBG_BUFF_UPDATE_TOREGEN_RAYS queue. */
	ccl_local_var unsigned int local_queue_atomics;
	if(ccl_local_thread_x == 0 && ccl_local_thread_y == 0) {
		local_queue_atomics = 0;
	}
	ccl_local_barrier();

	int ray_index = ccl_thread_y*ccl_size_x + ccl_thread_x;
	ray_index = get_ray_index(ray_index,
	                          QUEUE_ACTIVE_AND_REGENERATED_RAYS,
	                          Queue_data,
	                          queuesize,
	                          0);

	char enqueue_flag = (IS_STATE(ray_state, ray_index, RAY_TO_REGENERATE)) ? 1 : 0;
	enqueue_ray_index_local(ray_index,
		                    QUEUE_HITBG_BUFF_UPDATE_TOREGEN_RAYS,
		                    enqueue_flag,
		                    queuesize,
		                    &local_queue_atomics,
		                    Queue_data,
		                    Queue_index);

	/* TODO(lukas): Seems to conflict with queue code below? */
	if(ray_index != QUEUE_EMPTY_SLOT) {

		/* Continue on with shader evaluation. */

		kernel_shader_eval(NULL,
			               (ShaderData *)sd,
			               rng_coop,
			               Ray_coop,
			               PathState_coop,
			               Intersection_coop,
			               ray_state,
			               ray_index);
	}
}
