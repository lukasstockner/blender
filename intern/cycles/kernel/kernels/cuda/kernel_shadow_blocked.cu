#include "hip/hip_runtime.h"
/*
 * Copyright 2011-2015 Blender Foundation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#define __KERNEL_CUDA__
#define __KERNEL_CUDA_SPLIT__
#define __SPLIT_KERNEL__

#include "split/kernel_shadow_blocked.h"

extern "C" 
__global__ void kernel_cuda_path_trace_shadow_blocked(
        ccl_global char *sd_shadow,            /* Required for shadow blocked */
        ccl_global PathState *PathState_coop,  /* Required for shadow blocked */
        ccl_global Ray *LightRay_dl_coop,      /* Required for direct lighting's shadow blocked */
        ccl_global Ray *LightRay_ao_coop,      /* Required for AO's shadow blocked */
        Intersection *Intersection_coop_AO,
        Intersection *Intersection_coop_DL,
        ccl_global char *ray_state,
        ccl_global int *Queue_data,            /* Queue memory */
        ccl_global int *Queue_index,           /* Tracks the number of elements in each queue */
        int queuesize,                         /* Size (capacity) of each queue */
        int total_num_rays)
{

	int lidx = ccl_local_thread_y*ccl_local_size_x + ccl_local_thread_x;

	ccl_local_var unsigned int ao_queue_length;
	ccl_local_var unsigned int dl_queue_length;
	if(lidx == 0) {
		ao_queue_length = Queue_index[QUEUE_SHADOW_RAY_CAST_AO_RAYS];
		dl_queue_length = Queue_index[QUEUE_SHADOW_RAY_CAST_DL_RAYS];
	}
	ccl_local_barrier();

	/* flag determining if the current ray is to process shadow ray for AO or DL */
	char shadow_blocked_type = -1;

	int ray_index = QUEUE_EMPTY_SLOT;
	int thread_index = ccl_thread_y*ccl_size_x + ccl_thread_x;
	if(thread_index < ao_queue_length + dl_queue_length) {
		if(thread_index < ao_queue_length) {
			ray_index = get_ray_index(thread_index, QUEUE_SHADOW_RAY_CAST_AO_RAYS, Queue_data, queuesize, 1);
			shadow_blocked_type = RAY_SHADOW_RAY_CAST_AO;
		} else {
			ray_index = get_ray_index(thread_index - ao_queue_length, QUEUE_SHADOW_RAY_CAST_DL_RAYS, Queue_data, queuesize, 1);
			shadow_blocked_type = RAY_SHADOW_RAY_CAST_DL;
		}
	}

	if(ray_index == QUEUE_EMPTY_SLOT)
		return;

	kernel_shadow_blocked(NULL,
	                      (ShaderData *)sd_shadow,
	                      PathState_coop,
	                      LightRay_dl_coop,
	                      LightRay_ao_coop,
	                      Intersection_coop_AO,
	                      Intersection_coop_DL,
	                      ray_state,
	                      total_num_rays,
	                      shadow_blocked_type,
	                      ray_index);
}
