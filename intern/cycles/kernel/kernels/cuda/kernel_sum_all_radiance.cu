#include "hip/hip_runtime.h"
/*
 * Copyright 2011-2015 Blender Foundation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#define __KERNEL_CUDA__
#define __KERNEL_CUDA_SPLIT__
#define __SPLIT_KERNEL__

#include "split/kernel_split_common.h"
#include "split/kernel_sum_all_radiance.h"

extern "C" 
__global__ void kernel_cuda_path_trace_sum_all_radiance(
        ccl_global float *buffer,                    /* Output buffer of RenderTile */
        ccl_global float *per_sample_output_buffer,  /* Radiance contributed by all samples */
        int parallel_samples, int sw, int sh, int stride,
        int buffer_offset_x,
        int buffer_offset_y,
        int buffer_stride,
        int start_sample)
{
	kernel_sum_all_radiance(kernel_data.film.pass_stride,
	                        buffer,
	                        per_sample_output_buffer,
	                        parallel_samples,
	                        sw, sh, stride,
	                        buffer_offset_x,
	                        buffer_offset_y,
	                        buffer_stride,
	                        start_sample);
}
