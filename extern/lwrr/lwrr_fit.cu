#include "hip/hip_runtime.h"
/******************************************************************************\

  Copyright 2012 KAIST (Korea Advanced Institute of Science and Technology)
  All Rights Reserved.

  Permission to use, copy, modify and distribute this software and its
  documentation for educational, research and non-profit purposes, without
  fee, and without a written agreement is hereby granted, provided that the
  above copyright notice and the following three paragraphs appear in all
  copies. Any use in a commercial organization requires a separate license.

IN NO EVENT SHALL KAIST BE LIABLE TO ANY PARTY FOR DIRECT, INDIRECT, 
SPECIAL, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, INCLUDING LOST PROFITS, 
ARISING OUT OF THE USE OF THIS SOFTWARE AND ITS DOCUMENTATION, EVEN IF 
KAIST HAS BEEN ADVISED OF THE POSSIBILITY OF SUCH DAMAGES.

KAIST SPECIFICALLY DISCLAIM ANY WARRANTIES, INCLUDING, BUT NOT LIMITED TO, 
THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
PURPOSE.  THE SOFTWARE PROVIDED HEREUNDER IS ON AN "AS IS" BASIS, AND KAIST 
HAS NO OBLIGATION TO PROVIDE MAINTENANCE, SUPPORT, UPDATES, ENHANCEMENTS,
OR MODIFICATIONS.

   ---------------------------------
  |Please send all BUG REPORTS to:  |
  |                                 |
  |     moonbochang@gmail.com       |
  |                                 |
   ---------------------------------

  The authors may be contacted via:

Mail:         Bochang Moon or Sung-Eui Yoon
 			Dept. of Computer Science, E3-1 
KAIST 
291 Daehak-ro(373-1 Guseong-dong), Yuseong-gu 
DaeJeon, 305-701 
Republic of Korea
\*****************************************************************************/

#include "lwrr_fit.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

#include "svd.cuh"

#define PI		3.141592f	// Pi
#define PI2		6.283185f	// Pi^2

#define IMAD(a, b, c) ( __mul24((a), (b)) + (c) )

////////////////////////////////////////////////////////////////////////////////
// Global data handlers and parameters
////////////////////////////////////////////////////////////////////////////////
texture<float4, hipTextureType2D, hipReadModeElementType> g_img;
texture<float4, hipTextureType2D, hipReadModeElementType> g_texture;
texture<float4, hipTextureType2D, hipReadModeElementType> g_normal_depth;
texture<float,  hipTextureType2D, hipReadModeElementType> g_texture_moving;

texture<float4, hipTextureType2D, hipReadModeElementType> g_var_img;
texture<float4, hipTextureType2D, hipReadModeElementType> g_var_texture;
texture<float4, hipTextureType2D, hipReadModeElementType> g_var_texture_moving;
texture<float4, hipTextureType2D, hipReadModeElementType> g_var_normal;
texture<float4, hipTextureType2D, hipReadModeElementType> g_var_feature;

texture<float, hipTextureType2D, hipReadModeElementType> g_grey_texture;
texture<float, hipTextureType2D, hipReadModeElementType> g_depth;
texture<float, hipTextureType2D, hipReadModeElementType> g_var_depth;

texture<float, hipTextureType2D, hipReadModeElementType> g_spp;
texture<float, hipTextureType2D, hipReadModeElementType> g_spp_still;

//CUDA array descriptor
hipArray *g_src_img;
hipArray *g_src_var_img;
hipArray *g_src_texture;
hipArray *g_src_texture_moving;
hipArray *g_src_depth;
hipArray *g_src_normal_depth;
hipArray *g_src_var_feature;
hipArray *g_src_grey_texture;

hipArray *g_src_var_depth;
hipArray *g_src_var_texture;
hipArray *g_src_var_texture_moving;
hipArray *g_src_var_normal;
hipArray *g_src_spp;
hipArray *g_src_spp_still;

inline int iDivUp(int a, int b) { return ((a % b) != 0) ? (a / b + 1) : (a / b); }

__device__ float4 operator+ (const float4& a, const float4& b)  
{ 
	return make_float4(a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w); 
}

__device__ float4 operator- (const float4& a, const float4& b)  
{ 
	return make_float4(a.x - b.x, a.y - b.y, a.z - b.z, a.w - b.w); 
}

__device__ float dist2(const float4& val1, const float4& val2)
{
	return ((val1.x - val2.x) * (val1.x - val2.x) +
		    (val1.y - val2.y) * (val1.y - val2.y) +
		    (val1.z - val2.z) * (val1.z - val2.z));
}

__device__ float Color2Grey(const float4& color)
{
	return (color.x * 0.33333f + color.y * 0.33333f + color.z * 0.33333f);
}

// Input - A (only upper triangle is set!)
__device__ void cholesky(float *A, int n, float *L) 
{
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < (i+1); ++j) {
            float s = 0.0f;
            for (int k = 0; k < j; ++k)
                s += L[i * n + k] * L[j * n + k];
            L[i * n + j] = (i == j) ? sqrtf(A[i * n + i] - s) : (1.0f / L[j * n + j] * (A[j * n + i] - s));
        }
	}
}

// FilterMemoryCUDA members
void LWR_cuda_mem::allocMemory(int nPix)
{
	checkCudaErrors(hipMalloc((void **)&_d_out,      nPix * 3 * sizeof(float)));	

	checkCudaErrors(hipMalloc((void **)&_d_var_map,  nPix * sizeof(float)));
	checkCudaErrors(hipMalloc((void **)&_d_bias_map, nPix * sizeof(float)));

	checkCudaErrors(hipMalloc((void **)&_d_ranks, nPix * sizeof(float)));
	checkCudaErrors(hipMalloc((void **)&_d_hessians,  nDimens * nPix * sizeof(float)));		
	checkCudaErrors(hipMalloc((void **)&_d_transform, nDimens * nDimens * nPix * sizeof(float)));
	checkCudaErrors(hipMalloc((void **)&_d_bandwidth, nPix * sizeof(float)));

	checkCudaErrors(hipMalloc((void **)&_d_temp_mem1, nPix * 3 * sizeof(float)));
	checkCudaErrors(hipMalloc((void **)&_d_temp_mem2, nPix * 3 * sizeof(float)));

	checkCudaErrors(hipGetLastError());

	m_isInit = true;
}

void LWR_cuda_mem::deallocMemory()
{
	if (m_isInit) {
		checkCudaErrors(hipFree(_d_out));		
		checkCudaErrors(hipFree(_d_var_map));
		checkCudaErrors(hipFree(_d_bias_map));
		checkCudaErrors(hipFree(_d_ranks));
		checkCudaErrors(hipFree(_d_hessians));		
		checkCudaErrors(hipFree(_d_transform));		
		checkCudaErrors(hipFree(_d_bandwidth));

		checkCudaErrors(hipFree(_d_temp_mem1));	
		checkCudaErrors(hipFree(_d_temp_mem2));	

		checkCudaErrors(hipGetLastError());
	}
	m_isInit = false;
}

__device__ void getTransfCoord(float* iNewVec, float transf[][nDimens], float4& cNorDepth, float4& cTex,  
	                           int x, int y, int cx, int cy, int localD)
{	
	float4 iTex = tex2D(g_texture, x, y);
	float4 iNorDepth = tex2D(g_normal_depth, x, y);				
	for (int col = 0; col < localD; ++col) {							
		iNewVec[col] = transf[0][col] * (x - cx) + 
			           transf[1][col] * (y - cy) +
				       transf[2][col] * (iNorDepth.w - cNorDepth.w) +
					   transf[3][col] * (iNorDepth.x - cNorDepth.x) +
					   transf[4][col] * (iNorDepth.y - cNorDepth.y) +
					   transf[5][col] * (iNorDepth.z - cNorDepth.z) +
					   transf[6][col] * (iTex.x - cTex.x) + 
					   transf[7][col] * (iTex.y - cTex.y) +
					   transf[8][col] * (iTex.z - cTex.z);				
	}
}

__device__ void getTransfCoordExtended(float* iNewVec, float transf[][nDimens], float4& cNorDepth, float4& cTex,  float cMovTex,
	                           int x, int y, int cx, int cy, int localD)
{	
	float iMovTex = tex2D(g_texture_moving, x, y);
	float4 iTex = tex2D(g_texture, x, y);
	float4 iNorDepth = tex2D(g_normal_depth, x, y);			

	for (int col = 0; col < localD; ++col) {							
		iNewVec[col] = transf[0][col] * (x - cx) + 
			           transf[1][col] * (y - cy) +
				       transf[2][col] * (iNorDepth.w - cNorDepth.w) +
					   transf[3][col] * (iNorDepth.x - cNorDepth.x) +
					   transf[4][col] * (iNorDepth.y - cNorDepth.y) +
					   transf[5][col] * (iNorDepth.z - cNorDepth.z) +
					   transf[6][col] * (iTex.x - cTex.x) + 
					   transf[7][col] * (iTex.y - cTex.y) +
					   transf[8][col] * (iTex.z - cTex.z) +				
					   transf[9][col] * (iMovTex - cMovTex);
	}
}

__global__
void gaussian_fill_hole(const float* _in, const int* _spp, float* _out, int halfWidth, bool isColor, int xSize, int ySize)
{
	const int cx = IMAD(blockDim.x, blockIdx.x, threadIdx.x);
    const int cy = IMAD(blockDim.y, blockIdx.y, threadIdx.y);

	if (cx >= xSize || cy >= ySize)
		return;
	
	const int cIdx = cy * xSize + cx;	

	int ix, iy, idx;	

	int2 startWindow = make_int2(MAX(0, cx - halfWidth), MAX(0, cy - halfWidth));
	int2 endWindow   = make_int2(MIN(xSize - 1, cx + halfWidth), MIN(ySize - 1, cy + halfWidth));	

	int isHole = 0;

	isHole = isHole | (_spp[cIdx] == 0);

	if (isHole == 0) {
		if (isColor) {
			_out[cIdx * 3 + 0] = _in[cIdx * 3 + 0];
			_out[cIdx * 3 + 1] = _in[cIdx * 3 + 1];
			_out[cIdx * 3 + 2] = _in[cIdx * 3 + 2];
		}
		else
			_out[cIdx] = _in[cIdx];
		return;
	}


	float weight;
	float outColor[3] = {0.f,};
	float sumWeight = 0.f;
	for (iy = startWindow.y; iy <= endWindow.y; ++iy) {
		for (ix = startWindow.x; ix <= endWindow.x; ++ix) {	
			idx = iy * xSize + ix;
			if (_spp[idx] > 0) {
				weight = 1.f;
				if (isColor) {
					outColor[0] += weight * _in[idx * 3 + 0];
					outColor[1] += weight * _in[idx * 3 + 1];
					outColor[2] += weight * _in[idx * 3 + 2];
				}
				else 
					outColor[0] += weight * _in[idx];
				sumWeight += weight;
			}			
		}
	}

	float invSum = 1.f / max(sumWeight, 0.01f);
	if (isColor) {
		_out[cIdx * 3 + 0] = outColor[0] * invSum;
		_out[cIdx * 3 + 1] = outColor[1] * invSum;
		_out[cIdx * 3 + 2] = outColor[2] * invSum;
	}
	else
		_out[cIdx] = outColor[0] * invSum;
}

__global__
void gaussian_fit(float* _in, float* _out, float h, bool isColor, int xSize, int ySize, bool isIntegral)
{
	const int cx = IMAD(blockDim.x, blockIdx.x, threadIdx.x);
    const int cy = IMAD(blockDim.y, blockIdx.y, threadIdx.y);

	if (cx >= xSize || cy >= ySize)
		return;
	
	const int cIdx = cy * xSize + cx;	
	int halfWindowSize = int(h * 3.f + 0.5f);

	int ix, iy, idx;	

	int2 startWindow = make_int2(MAX(0, cx - halfWindowSize), MAX(0, cy - halfWindowSize));
	int2 endWindow   = make_int2(MIN(xSize - 1, cx + halfWindowSize), MIN(ySize - 1, cy + halfWindowSize));	

	float dist, weight;
	float outColor[3] = {0.f,};
	float sumWeight = 0.f;
	for (iy = startWindow.y; iy <= endWindow.y; ++iy) {
		for (ix = startWindow.x; ix <= endWindow.x; ++ix) {	
			idx = iy * xSize + ix;
			dist = (iy - cy) * (iy - cy) + (ix - cx) * (ix - cx);
			weight = expf(-1.f * dist / (2.f * h * h));
			if (isColor) {
				outColor[0] += weight * _in[idx * 3 + 0];
				outColor[1] += weight * _in[idx * 3 + 1];
				outColor[2] += weight * _in[idx * 3 + 2];
			}
			else 
				outColor[0] += weight * _in[idx];
			sumWeight += weight;
		}
	}

	float invSum = 1.f / sumWeight;
	if (isIntegral)
		invSum = 1.f;

	if (isColor) {
		_out[cIdx * 3 + 0] = outColor[0] * invSum;
		_out[cIdx * 3 + 1] = outColor[1] * invSum;
		_out[cIdx * 3 + 2] = outColor[2] * invSum;
	}
	else
		_out[cIdx] = outColor[0] * invSum;
}

__global__ void kernel_compute_derivatives_approx(float* _out,
												  float* _hessians, 
												  const float* _ranks, const float* _transform, 
												  const int xSize, const int ySize, const int MAX_HALF, const float h) 
{
	const int cx = IMAD(blockDim.x, blockIdx.x, threadIdx.x);
    const int cy = IMAD(blockDim.y, blockIdx.y, threadIdx.y);	

	// this branch should be here after shared memory loading!!
	if (cx >= xSize || cy >= ySize)
		return;

	const int cIdx = cy * xSize + cx;
	int2 startWindow = make_int2(MAX(0, cx - MAX_HALF), MAX(0, cy - MAX_HALF));
	int2 endWindow   = make_int2(MIN(xSize - 1, cx + MAX_HALF), MIN(ySize - 1, cy + MAX_HALF));	

	/////////////////////////////////////////////////
	// transform loading here
	const int localD = _ranks[cIdx];
	const int localQuadD = localD + localD;
	const int localP = localD + 1;	
	const int localQuadP = localQuadD + 1;

	float transform[nDimens][nDimens];

	#pragma unroll
	for (int row = 0; row < nDimens; ++row) {
		for (int col = 0; col < localD; ++col)
			transform[row][col] = _transform[xSize * ySize * (row * nDimens + col) + cIdx];
	}
	////////////////////////

	float4 cImg = tex2D(g_img, cx, cy);
	float4 cNorDepth = tex2D(g_normal_depth, cx, cy);
	float4 cTex = tex2D(g_texture, cx, cy);
	const int QuadP = (nDimens + 1) + nDimens + nDimens;
	float A[QuadP * QuadP] = {0.f,};	
	float XtB[QuadP][3] = {0.f,};	
	float iNewVecX[QuadP - 1];

#ifdef FEATURE_MOTION
	float cMovTex = tex2D(g_texture_moving, cx, cy);
#endif

	for (int y = startWindow.y; y <= endWindow.y; ++y) {		
		for (int x = startWindow.x; x <= endWindow.x; ++x) {
			float4 iImg = tex2D(g_img, x, y);	

#ifdef FEATURE_MOTION
			getTransfCoordExtended(iNewVecX, transform, cNorDepth, cTex, cMovTex, x, y, cx, cy, localD);						
#else
			getTransfCoord(iNewVecX, transform, cNorDepth, cTex, x, y, cx, cy, localD);			
#endif
			
			float weight = 1.f;
			for (int col = 0; col < localD; ++col) {
				float t = iNewVecX[col] / h;
				if (fabs(t) < 1.f)
					weight *= 0.75f * (1.f - t * t); 
				else {
					weight = 0.f;
					break;
				}
			}	

			if (weight > 0.f) {				
#ifdef OUTLIER_TRICK	
				weight /= MAX(iImg.w, 1.f);							
#endif
			
				for (int col = 0; col < localD; ++col)
					iNewVecX[localD + col] = iNewVecX[col] * iNewVecX[col];		

				A[0] += weight;				
				XtB[0][0] += weight * iImg.x;
				XtB[0][1] += weight * iImg.y;
				XtB[0][2] += weight * iImg.z;				

				
				for (int col = 1; col < localQuadP; ++col) {
					float temp = weight * iNewVecX[col - 1];

					A[col] += temp;					
					XtB[col][0] += temp * iImg.x;
					XtB[col][1] += temp * iImg.y;
					XtB[col][2] += temp * iImg.z;
				}

				// other rows
				#pragma unroll
				for (int row = 1; row < localQuadP; ++row) {
					for (int col = row; col < localQuadP; ++col) 
						A[row * localQuadP + col] += weight * iNewVecX[row - 1] * iNewVecX[col - 1];
				}		
			}
		}
	}

	for (int row = 0; row < localQuadP; ++row)
		A[row * localQuadP + row] += 0.001f;

	float* L = A;
	cholesky(A, localQuadP, L);

	for (int c = 0; c < 3; ++c) {
		////////////////////////////////////////////////////////////////////////////////////////////////
		// Forward substitution
		XtB[0][c] = XtB[0][c] / L[0 * localQuadP + 0];
		for (int i = 1; i < localQuadP; ++i) {
			float s = 0.f;
			for (int k = 0; k < i; ++k) {
				s += L[i * localQuadP + k] * XtB[k][c];
			}
			XtB[i][c] = (XtB[i][c] - s) / L[i * localQuadP + i];
		}
		////////////////////////////////////////////////////////////////////////////////////////////////
		// Backward substituation
		XtB[localQuadP - 1][c] = XtB[localQuadP - 1][c] / L[(localQuadP - 1) * localQuadP + (localQuadP - 1)];		
		for (int i = localQuadP - 2; i >= 0; --i) {
			float s = 0.f;
			for (int k = localQuadP - 1; k > i; --k) {
				s += L[k * localQuadP + i] * XtB[k][c];
			}
			XtB[i][c] = (XtB[i][c] - s) / L[i * localQuadP + i];
		}
	}
	
	for (int f = 0; f < nDimens; ++f) { 			
		_hessians[xSize * ySize * f + cIdx] = 2.f * (fabs(XtB[localP + f][0]) * 0.33333f +
													 fabs(XtB[localP + f][1]) * 0.33333f +
													 fabs(XtB[localP + f][2]) * 0.33333f);
	}
}

__global__ void kernel_compute_transform(float* _out, float* _transform, float* _ranks,
										 const int xSize, const int ySize, const int MAX_HALF, const float* _bandwidth)
{
	const int cx = IMAD(blockDim.x, blockIdx.x, threadIdx.x);
    const int cy = IMAD(blockDim.y, blockIdx.y, threadIdx.y);		
	if (cx >= xSize || cy >= ySize)
		return;

	int2 startWindow = make_int2(MAX(0, cx - MAX_HALF), MAX(0, cy - MAX_HALF));
	int2 endWindow   = make_int2(MIN(xSize - 1, cx + MAX_HALF), MIN(ySize - 1, cy + MAX_HALF));	

	float4 avgNorDepth = make_float4(0.f, 0.f, 0.f, 0.f);
	float4 avgTex = make_float4(0.f, 0.f, 0.f, 0.f);
	float avgMovTex = 0.f;

	for (int y = endWindow.y; y >= startWindow.y; y--) {		
		for (int x = endWindow.x; x >= startWindow.x; x--) {
			const float4& iTex = tex2D(g_texture, x, y);
			const float4& iNorDepth = tex2D(g_normal_depth, x, y);
			avgNorDepth = avgNorDepth + iNorDepth;
			avgTex = avgTex + iTex;

#ifdef FEATURE_MOTION
			avgMovTex += tex2D(g_texture_moving, x, y);
#endif
		}
	}

	float invN = 1.f / ((endWindow.y - startWindow.y + 1) * (endWindow.x - startWindow.x + 1));
	avgNorDepth = make_float4(avgNorDepth.x * invN, avgNorDepth.y * invN, avgNorDepth.z * invN, avgNorDepth.w * invN);
	avgTex = make_float4(avgTex.x * invN, avgTex.y * invN, avgTex.z * invN, 0.f);

#ifdef FEATURE_MOTION
	avgMovTex *= invN;
#endif

	/////////////////////////////////////////////////////////////////////////////////
	// Column normalization
	float factorDepth, factorNormal, factorTexture;
	factorDepth = factorNormal = factorTexture = 0.f;

#ifndef FEATURE_MOTION
	{
		for (int y = startWindow.y; y <= endWindow.y; ++y) {		
			for (int x = startWindow.x; x <= endWindow.x; ++x) {
				const float4& iNorDepth = tex2D(g_normal_depth, x, y);
				const float4& iTex = tex2D(g_texture, x, y);
				factorDepth = max(factorDepth, fabs(iNorDepth.w - avgNorDepth.w));
				factorNormal = max(factorNormal, dist2(iNorDepth, avgNorDepth));
				factorTexture = max(factorTexture, dist2(iTex, avgTex));
			}
		}
		factorTexture = 1.f / max(sqrtf(factorTexture), 0.01f);
		factorNormal = 1.f / max(sqrtf(factorNormal), 0.01f);	
		factorDepth = 1.f / max(factorDepth, 0.01f);
	}
#else
	float facMovTex = 0.f;	
	for (int y = startWindow.y; y <= endWindow.y; ++y) {		
		for (int x = startWindow.x; x <= endWindow.x; ++x) {			
			const float iMovTex = tex2D(g_texture_moving, x, y);
			facMovTex = max(facMovTex, fabs(iMovTex - avgMovTex));
		}
	}
	facMovTex = 1.f / max(sqrtf(facMovTex), 0.01f);		

	float spp_still = tex2D(g_spp_still, cx, cy);
	if (spp_still > 0.01f) 
	{
		for (int y = startWindow.y; y <= endWindow.y; ++y) {		
			for (int x = startWindow.x; x <= endWindow.x; ++x) {
				const float4& iNorDepth = tex2D(g_normal_depth, x, y);
				const float4& iTex = tex2D(g_texture, x, y);
				factorDepth = max(factorDepth, fabs(iNorDepth.w - avgNorDepth.w));
				factorNormal = max(factorNormal, dist2(iNorDepth, avgNorDepth));
				factorTexture = max(factorTexture, dist2(iTex, avgTex));
			}
		}
		factorTexture = 1.f / max(sqrtf(factorTexture), 0.01f);
		factorDepth = 1.f / max(factorDepth, 0.01f);
		factorNormal = 1.f / max(sqrtf(factorNormal), 0.01f);	
	}	
	// else
	// We don't have any samples for this basic G-buffer (Undefined) - Ignore those buffers by making factor zero
#endif
	
	float delta[nDimens];
	float A[nDimens * nDimens] = {0.f,};	
	float errNorm = 0.f;	
	const float invMaxHalfWindow = 1.f / MAX_HALF;

	for (int y = endWindow.y; y >= startWindow.y; y--) {		
		for (int x = endWindow.x; x >= startWindow.x; x--) {
			const float4& iTex = tex2D(g_texture, x, y);
			const float4& iNorDepth = tex2D(g_normal_depth, x, y);

			delta[0] = (x - cx) * invMaxHalfWindow;
			delta[1] = (y - cy) * invMaxHalfWindow;
			delta[2] = (iNorDepth.w - avgNorDepth.w) * factorDepth;
			delta[3] = (iNorDepth.x - avgNorDepth.x) * factorNormal;
			delta[4] = (iNorDepth.y - avgNorDepth.y) * factorNormal;
			delta[5] = (iNorDepth.z - avgNorDepth.z) * factorNormal;
			delta[6] = (iTex.x - avgTex.x) * factorTexture;
			delta[7] = (iTex.y - avgTex.y) * factorTexture;
			delta[8] = (iTex.z - avgTex.z) * factorTexture;

#ifdef FEATURE_MOTION
			delta[9] = (tex2D(g_texture_moving, x, y) - avgMovTex) * facMovTex;
#endif

			#pragma unroll
			for (int row = 0; row < nDimens; ++row) {				
				for (int col = row; col < nDimens; ++col) 
					A[row * nDimens + col] += delta[row] * delta[col];				
			}	

			const float4& varFeature = tex2D(g_var_feature, x, y);		
			errNorm += (factorDepth * factorDepth * varFeature.x + 
		                factorNormal * factorNormal * varFeature.y +  
					    factorTexture * factorTexture * varFeature.z);		
#ifdef FEATURE_MOTION
			errNorm += facMovTex * facMovTex * varFeature.w;
#endif
		}
	}	

	// Fill lower parts of A
	for (int row = 1; row < nDimens; ++row) {
		for (int col = 0; col < row; ++col) {
			A[row * nDimens + col] = A[col * nDimens + row];
		}
	}

	float V[nDimens * nDimens] = {0.f,};
	float S[nDimens];

	// Initialize V as an identity 
	for (int col = 0; col < nDimens; col++) 
		V[col * nDimens + col] = 1.f;
	
	int rank = svd(A, V, S, nDimens);	
	for (int col = 0; col < nDimens; ++col)
		S[col] = sqrtf(fabs(S[col]));

	// We approximate the Spectral norm (expensive) using the Frobenius norm (cheap) 
	errNorm = sqrtf(errNorm) / (sqrtf(rank) * 0.5f);	
	rank = 0;
	
	// 0.01 is for thin-SVD 
	float tol = 0.01f + 2.f * errNorm;																					

	// Update V = VS^(-2)Vt		
	// T-SVD
	const int nPix = xSize * ySize;
	const int cIdx = cy * xSize + cx;

	for (int col = 0; col < nDimens; ++col) {	
		float singular = sqrtf(fabs(S[col]));					

		if (singular > tol || col < 2) {		
			++rank;
			// Pre-Multiply normalization factor
			_transform[nPix * (0 * nDimens + col) + cIdx] = V[0 * nDimens + col] * invMaxHalfWindow;
			_transform[nPix * (1 * nDimens + col) + cIdx] = V[1 * nDimens + col] * invMaxHalfWindow;
			_transform[nPix * (2 * nDimens + col) + cIdx] = V[2 * nDimens + col] * factorDepth;
			_transform[nPix * (3 * nDimens + col) + cIdx] = V[3 * nDimens + col] * factorNormal;
			_transform[nPix * (4 * nDimens + col) + cIdx] = V[4 * nDimens + col] * factorNormal;
			_transform[nPix * (5 * nDimens + col) + cIdx] = V[5 * nDimens + col] * factorNormal;
			_transform[nPix * (6 * nDimens + col) + cIdx] = V[6 * nDimens + col] * factorTexture;		
			_transform[nPix * (7 * nDimens + col) + cIdx] = V[7 * nDimens + col] * factorTexture;		
			_transform[nPix * (8 * nDimens + col) + cIdx] = V[8 * nDimens + col] * factorTexture;	

#ifdef FEATURE_MOTION
			_transform[nPix * (9 * nDimens + col) + cIdx] = V[9 * nDimens + col] * facMovTex;	
#endif
		}
	}	
	_ranks[cIdx] = rank;		
}

__global__ void kernel_fit_anisotropic(float* _out, 
									   const float* _ranks, const float* _transform,
									   float* _hessians, 
									   int xSize, int ySize, const int MAX_HALF, const float* _bandwidth,
									   float* _bias_map, float* _var_map, bool isFinalFit) 
{
	const int cx = IMAD(blockDim.x, blockIdx.x, threadIdx.x);
    const int cy = IMAD(blockDim.y, blockIdx.y, threadIdx.y);	

	// this branch should be here after shared memory loading!!
	if (cx >= xSize || cy >= ySize)
		return;

	int2 startWindow = make_int2(MAX(0, cx - MAX_HALF), MAX(0, cy - MAX_HALF));
	int2 endWindow   = make_int2(MIN(xSize - 1, cx + MAX_HALF), MIN(ySize - 1, cy + MAX_HALF));	

	const int nPix = xSize * ySize;
	const int cIdx = cy * xSize + cx;
	const int localD = _ranks[cIdx];
	const int localP = localD + 1;	

	float transform[nDimens][nDimens];
	for (int row = 0; row < nDimens; ++row) {
		for (int col = 0; col < localD; ++col) {			
			transform[row][col] = _transform[xSize * ySize * (row * nDimens + col) + cIdx];
		}
	}

	//
	float band[nDimens];
	for (int i = 0; i < localD; ++i) {
		// feature bandwidth bi = (0, 2.5]
		float bi = 1.f / sqrtf(fabs(_hessians[nPix * i + cIdx]) + 0.16f);				
		band[i] = _bandwidth[cIdx] * bi;				
	}

	float iNewVecX[nDimens];	

	float4 cImg = tex2D(g_img, cx, cy);	
	float cGreyImg = Color2Grey(cImg);

	float4 cVarImg = tex2D(g_var_img, cx, cy);
	float4 cNorDepth = tex2D(g_normal_depth, cx, cy);
	float4 cTex = tex2D(g_texture, cx, cy);

#ifdef FEATURE_MOTION
	float cMovTex = tex2D(g_texture_moving, cx, cy);
#endif

	float __weights[MAX_HALF_WINDOW * 2 + 1][MAX_HALF_WINDOW * 2 + 1];
	float A[(nDimens + 1) * (nDimens + 1)] = {0.f,};		
	int nSample = 0;
	float sumWeight = 0.f;
	for (int y = startWindow.y; y <= endWindow.y; ++y) {		
		for (int x = startWindow.x; x <= endWindow.x; ++x) {
			const float4& iImg = tex2D(g_img, x, y);	

			// Confidence interval test: reject stastically distant samples
			float iGreyImg = Color2Grey(iImg);			
			if (fabs(cGreyImg - iGreyImg) > 3.f * (sqrtf(iImg.w) + sqrtf(cImg.w)) + 0.005f) {
				__weights[y - startWindow.y][x - startWindow.x] = 0.f;
				continue;		
			}			

#ifdef FEATURE_MOTION
			getTransfCoordExtended(iNewVecX, transform, cNorDepth, cTex, cMovTex, x, y, cx, cy, localD);				
#else
			getTransfCoord(iNewVecX, transform, cNorDepth, cTex, x, y, cx, cy, localD);	
#endif

			float weight = 1.f;
			for (int col = 0; col < localD; ++col) {
				float t = iNewVecX[col] / band[col];
				if (fabs(t) < 1.f)
					weight *= 0.75f * (1.f - t * t); 
				else {
					weight = 0.f;
					break;
				}
			}			

			if (weight > 0.0f) {				
#ifdef OUTLIER_TRICK
				// outlier	
				weight /= MAX(iImg.w, 1.0f);		
#endif
				A[0] += weight;
				for (int col = 1; col < localP; ++col)									
					A[col] += weight * iNewVecX[col - 1];					

				#pragma unroll
				for (int row = 1; row < localP; ++row) {
					for (int col = row; col < localP; ++col)												
						A[row*localP+col] += weight * iNewVecX[row - 1] * iNewVecX[col - 1];						
				}
				++nSample;
			}
			__weights[y - startWindow.y][x - startWindow.x] = weight;
			sumWeight += weight;
		}			
	}
	if (nSample <= 1) {	
		_out[cIdx * 3 + 0] = cImg.x;
		_out[cIdx * 3 + 1] = cImg.y;
		_out[cIdx * 3 + 2] = cImg.z;
		_bias_map[cIdx] = 0.f;				
		_var_map[cIdx] = cImg.w;		
		return;
	}

	for (int row = 0; row < localP; ++row)
		A[row * localP + row] += 0.0001f;

	float* L = A;
	cholesky(A, localP, L);

	// invL^T = save L^-1 to upper part in L
	float invL[(nDimens + 1) * (nDimens + 1)] = {0.f,};
	for (int j = localP - 1; j >= 0; --j) {
		invL[j * localP + j] = 1.f / L[j * localP + j];
		for (int k = j + 1; k < localP; ++k) {
			for (int i = j + 1; i < localP; ++i) {
				invL[k * localP + j] += invL[k * localP + i] * L[i * localP + j];
			}
		}
		for (int k = j + 1; k < localP; ++k) {
			invL[k * localP + j] = -1.f * invL[j * localP + j] * invL[k * localP + j];
		}
	}

	// First row in (XtWX)^-1
	float invA[nDimens + 1];
	for (int i = 0; i < localP; ++i) {
		float e = 0.f;
		for (int k = i; k < localP; ++k) {
			e += invL[k * localP] * invL[k * localP + i];
		}
		invA[i] = e;
	}

	// Final Fitting	
	float err_beta[3] = {0.f,};	
	float err_sum_l = 0.f;
	float beta[3] = {0.f,};

	float var = 0.f;
	float err_var = 0.f;

	for (int y = startWindow.y; y <= endWindow.y; ++y) {		
		for (int x = startWindow.x; x <= endWindow.x; ++x) {						
			float weight = __weights[y - startWindow.y][x - startWindow.x];

			if (weight > 0.f) {
#ifdef FEATURE_MOTION
				getTransfCoordExtended(iNewVecX, transform, cNorDepth, cTex, cMovTex, x, y, cx, cy, localD);					
#else
				getTransfCoord(iNewVecX, transform, cNorDepth, cTex, x, y, cx, cy, localD);					
#endif

				float l = invA[0];

				for (int f = 0; f < localD; ++f)											
					l += iNewVecX[f] * invA[f + 1];			

				l *= weight;

				const float4& iImg = tex2D(g_img, x, y);				

				beta[0] += l * iImg.x;
				beta[1] += l * iImg.y;
				beta[2] += l * iImg.z;

				var += l * l * iImg.w;

				//
				if (l > 0.f) {
					err_beta[0] += l * iImg.x;
					err_beta[1] += l * iImg.y;
					err_beta[2] += l * iImg.z;

					err_var += l * l * iImg.w;
					err_sum_l += l;
				}								
			}
		}
	}

	// exception handling
	if (beta[0] < 0.f || beta[1] < 0.f || beta[2] < 0.f) {	
		err_sum_l = max(err_sum_l, 0.001f);
		beta[0] = err_beta[0] / err_sum_l;
		beta[1] = err_beta[1] / err_sum_l;
		beta[2] = err_beta[2] / err_sum_l;

		var = err_var / (err_sum_l * err_sum_l);
	}


	_out[cIdx * 3 + 0] = max(0.f, beta[0]);
	_out[cIdx * 3 + 1] = max(0.f, beta[1]);
	_out[cIdx * 3 + 2] = max(0.f, beta[2]);	

	// Store bias and variance from this fitting!
	_bias_map[cIdx] = (beta[0] - cImg.x) * 0.33333f + 
		              (beta[1] - cImg.y) * 0.33333f + 
					  (beta[2] - cImg.z) * 0.33333f;	
	_var_map[cIdx] = var;		
}

extern "C"
void allocTextureMemory(int xSize, int ySize) 
{
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();	
	hipChannelFormatDesc channelDescGrey = hipCreateChannelDesc<float>();	

	checkCudaErrors(hipMallocArray(&g_src_texture, &channelDesc, xSize, ySize));
	checkCudaErrors(hipMallocArray(&g_src_img, &channelDesc, xSize, ySize));	
	checkCudaErrors(hipMallocArray(&g_src_var_feature, &channelDesc, xSize, ySize));			
	checkCudaErrors(hipMallocArray(&g_src_normal_depth, &channelDesc, xSize, ySize));	

#ifdef FEATURE_MOTION
	checkCudaErrors(hipMallocArray(&g_src_spp_still, &channelDescGrey, xSize, ySize));	
	checkCudaErrors(hipMallocArray(&g_src_texture_moving, &channelDescGrey, xSize, ySize));	
#endif

	checkCudaErrors(hipGetLastError());
}

extern "C"
void initDeviceMemory(const float* _img, const float* _var_img, const float* _texture, const float* _var_texture, 
                      const float* _normal, const float* _var_normal, const float* _depth, const float* _var_depth, 
					  const float* _texture_moving, const float* _var_texture_moving,
					  const int* _mapSPP, int xSize, int ySize,
					  const int* _mapMovingSPP)
{
	int nPix = xSize * ySize;

	float4* _h_texture = (float4*)malloc(nPix * sizeof(float4));
	float4* _h_img = (float4*)malloc(nPix * sizeof(float4));
	float4* _h_normal_depth = (float4*)malloc(nPix * sizeof(float4));
	float4* _h_var_feature = (float4*)malloc(nPix * sizeof(float4));

#ifdef FEATURE_MOTION
	float* _h_texture_moving = (float*)malloc(nPix * sizeof(float));
	float* _h_spp_still = (float*)malloc(nPix * sizeof(float));
#endif

#pragma omp parallel for schedule(guided, 4)
	for (int i = 0; i < nPix; ++i) {
		float inv_spp_idx = 1.f / (float)_mapSPP[i];
		float mean_var_img = max(0.f, inv_spp_idx * (_var_img[i * 3 + 0] * 0.33333f +
											         _var_img[i * 3 + 1] * 0.33333f +
											         _var_img[i * 3 + 2] * 0.33333f));
		float mean_var_tex = max(0.f, inv_spp_idx * (_var_texture[i * 3 + 0] +
										       	     _var_texture[i * 3 + 1] +
											         _var_texture[i * 3 + 2]));
		float mean_var_nor = max(0.f, inv_spp_idx * (_var_normal[i * 3 + 0] + _var_normal[i * 3 + 1] + _var_normal[i * 3 + 2]));			               
		float mean_var_depth = max(0.f, inv_spp_idx * _var_depth[i]);

		_h_img[i] = make_float4(_img[i * 3 + 0], _img[i * 3 + 1], _img[i * 3 + 2], mean_var_img);		
		_h_texture[i] = make_float4(_texture[i * 3 + 0], _texture[i * 3 + 1], _texture[i * 3 + 2], 0.f);
		_h_normal_depth[i] = make_float4(_normal[i * 3 + 0], _normal[i * 3 + 1], _normal[i * 3 + 2], _depth[i]);


#ifndef FEATURE_MOTION
		_h_var_feature[i] = make_float4(mean_var_depth, mean_var_nor, mean_var_tex, 0.f);
#else
		float mean_var_moving = inv_spp_idx * (_var_texture_moving[i * 3 + 0] + 
			                                   _var_texture_moving[i * 3 + 1] +
											   _var_texture_moving[i * 3 + 2] );
		_h_var_feature[i] = make_float4(mean_var_depth, mean_var_nor, mean_var_tex, mean_var_moving);
		_h_texture_moving[i] = _texture_moving[i * 3 + 0] * 0.33333f + 
			                   _texture_moving[i * 3 + 1] * 0.33333f +
							   _texture_moving[i * 3 + 2] * 0.33333f;
		_h_spp_still[i] = _mapSPP[i] - _mapMovingSPP[i];
#endif
	}	    

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();	

	checkCudaErrors(hipMemcpyToArray(g_src_texture, 0, 0,		_h_texture,      nPix * sizeof(float4), hipMemcpyHostToDevice));	
	checkCudaErrors(hipMemcpyToArray(g_src_img, 0, 0,          _h_img,          nPix * sizeof(float4), hipMemcpyHostToDevice));	
	checkCudaErrors(hipMemcpyToArray(g_src_var_feature, 0, 0,  _h_var_feature,  nPix * sizeof(float4), hipMemcpyHostToDevice));	
	checkCudaErrors(hipMemcpyToArray(g_src_normal_depth, 0, 0, _h_normal_depth, nPix * sizeof(float4), hipMemcpyHostToDevice));	


	checkCudaErrors(hipBindTextureToArray(&g_texture, g_src_texture, &channelDesc));
	checkCudaErrors(hipBindTextureToArray(&g_img, g_src_img, &channelDesc));	
	checkCudaErrors(hipBindTextureToArray(&g_var_feature, g_src_var_feature, &channelDesc));	
	checkCudaErrors(hipBindTextureToArray(&g_normal_depth, g_src_normal_depth, &channelDesc));

#ifdef FEATURE_MOTION
	hipChannelFormatDesc channelDescGrey = hipCreateChannelDesc<float>();

	checkCudaErrors(hipMemcpyToArray(g_src_spp_still, 0, 0, _h_spp_still, nPix * sizeof(float), hipMemcpyHostToDevice));	
	checkCudaErrors(hipMemcpyToArray(g_src_texture_moving, 0, 0, _h_texture_moving, nPix * sizeof(float), hipMemcpyHostToDevice));	

	checkCudaErrors(hipBindTextureToArray(&g_spp_still, g_src_spp_still, &channelDescGrey));
	checkCudaErrors(hipBindTextureToArray(&g_texture_moving, g_src_texture_moving, &channelDescGrey));
#endif

	checkCudaErrors(hipGetLastError());	

	free(_h_texture);
	free(_h_normal_depth);
	free(_h_img);		
	free(_h_var_feature);

#ifdef FEATURE_MOTION
	free(_h_texture_moving);
	free(_h_spp_still);
#endif
}

extern "C"
void freeDeviceMemory()
{
	checkCudaErrors(hipFreeArray(g_src_texture));	
	checkCudaErrors(hipFreeArray(g_src_img));	
	checkCudaErrors(hipFreeArray(g_src_var_feature));	
	checkCudaErrors(hipFreeArray(g_src_normal_depth));	

#ifdef FEATURE_MOTION
	checkCudaErrors(hipFreeArray(g_src_spp_still));	
	checkCudaErrors(hipFreeArray(g_src_texture_moving));	
#endif
	checkCudaErrors(hipGetLastError());	
}

extern "C"
void localFitShared(float* _dbgImg,
					int xSize, int ySize, const int MAX_HALF,
					float* _dbg_hessians,
					float** _fit_map, float** _var_map, float** _bias_map, float* _ranks,
					float** _width_guess,
					LWR_cuda_mem& gloMemory,
					const int* _spp) 
{
	hipDeviceSetCacheConfig(hipFuncCachePreferL1);

	int nPix = xSize * ySize;
	dim3 threads(BLOCKDIM, BLOCKDIM);
    dim3 grid(iDivUp(xSize, BLOCKDIM), iDivUp(ySize, BLOCKDIM));

	// init cudaPrintf
	hipEvent_t start, stop;	
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float time;

	///////////////////////////////////////
	hipEventRecord(start, 0);
	kernel_compute_transform<<<grid, threads>>>(gloMemory._d_out, gloMemory._d_transform, gloMemory._d_ranks,
												xSize, ySize, MAX_HALF, gloMemory._d_bandwidth);
	checkCudaErrors(hipDeviceSynchronize());
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);		
	hipEventElapsedTime(&time, start, stop);
	//
	printf("Time for Computing Transform Matrix: %f ms\n", time);	
	checkCudaErrors(hipMemcpy(_ranks, gloMemory._d_ranks, nPix * sizeof(float), hipMemcpyDeviceToHost));	

	/////////////////////	
	hipEventRecord(start, 0);	
	const float band_derivatives = 1.f;
	kernel_compute_derivatives_approx<<<grid, threads>>>(gloMemory._d_out,
													     gloMemory._d_hessians, 
														 gloMemory._d_ranks, gloMemory._d_transform, 
														 xSize, ySize, MAX_HALF, band_derivatives);		
	checkCudaErrors(hipDeviceSynchronize());
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);		
	hipEventElapsedTime(&time, start, stop);
	printf("Time for derivatives: %f ms\n", time);
	
	for (int iter = 0; iter < NUM_TEST; ++iter) {		
		hipEventRecord(start, 0);	

		checkCudaErrors(hipMemcpy(gloMemory._d_bandwidth, _width_guess[iter], nPix * sizeof(float), hipMemcpyHostToDevice));
		//
		kernel_fit_anisotropic<<<grid, threads>>>(gloMemory._d_out, 
												  gloMemory._d_ranks, gloMemory._d_transform,						
												  gloMemory._d_hessians, xSize, ySize, MAX_HALF, gloMemory._d_bandwidth,
												  gloMemory._d_bias_map, gloMemory._d_var_map, false);				
		checkCudaErrors(hipDeviceSynchronize());
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);		
		hipEventElapsedTime(&time, start, stop);
		printf("Time for the anisotropic kernel: %f ms\n", time);

		checkCudaErrors(hipMemcpy(_var_map[iter], gloMemory._d_var_map, nPix * sizeof(float), hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(_bias_map[iter], gloMemory._d_bias_map, nPix * sizeof(float), hipMemcpyDeviceToHost));
	}
	/////////////////////////////////
	
	checkCudaErrors(hipGetLastError());	
}

extern "C"
void localFitSharedFinal(float* _out, 
						 int xSize, int ySize, const int MAX_HALF,	
						 float* _bandwidth,
						 LWR_cuda_mem& gloMemory,
						 float* _opt_var) 
{
	int nPix = xSize * ySize;

	dim3 threads(BLOCKDIM, BLOCKDIM);
    dim3 grid(iDivUp(xSize, BLOCKDIM), iDivUp(ySize, BLOCKDIM));

	// init cudaPrintf
	hipEvent_t start, stop;	
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float time;

	checkCudaErrors(hipMemcpy(gloMemory._d_bandwidth, _bandwidth, nPix * sizeof(float), hipMemcpyHostToDevice));

	hipEventRecord(start, 0);			

	kernel_fit_anisotropic<<<grid, threads>>>(gloMemory._d_out, 
											  gloMemory._d_ranks, gloMemory._d_transform, 
											  gloMemory._d_hessians, xSize, ySize, MAX_HALF, gloMemory._d_bandwidth,
											  gloMemory._d_bias_map, gloMemory._d_var_map, true);
	checkCudaErrors(hipDeviceSynchronize());
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);		
	hipEventElapsedTime(&time, start, stop);
	printf("Time for the adaptive kernel: %f ms\n", time);

	///////////////////////////////////////////////////////////////////////////////
	// data transfer from device to host
	checkCudaErrors(hipMemcpy(_out, gloMemory._d_out, nPix * 3 * sizeof(float), hipMemcpyDeviceToHost));

	// free texture memory
	checkCudaErrors(hipUnbindTexture(&g_texture));	
	checkCudaErrors(hipUnbindTexture(&g_img));	
	checkCudaErrors(hipUnbindTexture(&g_var_feature));	
	checkCudaErrors(hipUnbindTexture(&g_normal_depth));	

#ifdef FEATURE_MOTION
	checkCudaErrors(hipUnbindTexture(&g_spp_still));	
	checkCudaErrors(hipUnbindTexture(&g_texture_moving));	
#endif

	checkCudaErrors(hipGetLastError());	
}

extern "C"
void localGuassian2(float* _img, float* _d_in_mem, float* _d_out_mem, int xSize, int ySize, float h, bool isColor, bool isIntegral) 
{
	int nPix = xSize * ySize;
	size_t memSize = nPix * sizeof(float);
	if (isColor)
		memSize *= 3;
	/////////////////////////////////////////////////////////////////////////////////

	/////////////////////////////////////////////////////////////////////////////////
	// data transfer from host to device	
	checkCudaErrors(hipMemcpy(_d_in_mem, _img, memSize, hipMemcpyHostToDevice));

	///////////////////////////////////////////////////////////////////////////////
	// launch CUDA kernel
	dim3 threads(BLOCKDIM, BLOCKDIM);
    dim3 grid(iDivUp(xSize, BLOCKDIM), iDivUp(ySize, BLOCKDIM));

	gaussian_fit<<<grid, threads>>>(_d_in_mem, _d_out_mem, h, isColor, xSize, ySize, isIntegral);
	checkCudaErrors(hipDeviceSynchronize());			

	checkCudaErrors(hipMemcpy(_img, _d_out_mem, memSize, hipMemcpyDeviceToHost));
	checkCudaErrors(hipGetLastError());	
	/////////////////////////////////////////////////////////////////////////////////
}

extern "C"
void localGuassianFillHoles(float* _img, const int* _still_spp, int xSize, int ySize, int halfWidth, bool isColor) 
{
	int nPix = xSize * ySize;
	size_t memSize = nPix * sizeof(float);
	if (isColor)
		memSize *= 3;

	/////////////////////////////////////////////////////////////////////////////////
	// allocate the device memory	
	float *_d_in = NULL;
	float *_d_out = NULL;
	int *_d_spp = NULL;

	/////////////////////////////////////////////////////////////////
	// device memory allocation	
	checkCudaErrors(hipMalloc((void **)&_d_in, memSize));
	checkCudaErrors(hipMalloc((void **)&_d_out, memSize));
	checkCudaErrors(hipMalloc((void **)&_d_spp, nPix * sizeof(int)));
	/////////////////////////////////////////////////////////////////////////////////

	/////////////////////////////////////////////////////////////////////////////////
	// data transfer from host to device	
	checkCudaErrors(hipMemcpy(_d_in, _img, memSize, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(_d_spp, _still_spp, nPix * sizeof(int), hipMemcpyHostToDevice));

	///////////////////////////////////////////////////////////////////////////////
	// launch CUDA kernel
	dim3 threads(BLOCKDIM, BLOCKDIM);
    dim3 grid(iDivUp(xSize, BLOCKDIM), iDivUp(ySize, BLOCKDIM));

	gaussian_fill_hole<<<grid, threads>>>(_d_in, _d_spp, _d_out, halfWidth, isColor, xSize, ySize);
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipGetLastError());			

	///////////////////////////////////////////////////////////////////////////////
	// data transfer from device to host
	checkCudaErrors(hipMemcpy(_img, _d_out, memSize, hipMemcpyDeviceToHost));

	/////////////////////////////////////////////////////////////////////////////////
	// free device memory	
	checkCudaErrors(hipFree(_d_in));
	checkCudaErrors(hipFree(_d_out));	
	checkCudaErrors(hipFree(_d_spp));		
	/////////////////////////////////////////////////////////////////////////////////
}
